#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>

int main(){
	hipDeviceProp_t prop;  // Object that has access to the gpus traits
	
	int count; 
	hipGetDeviceCount(&count);
	
	for (int i = 0; i < count; i++){
		hipGetDeviceProperties(&prop, i);
		printf("name %s \n", prop.name);
		printf("clockrate %i \n", prop.clockRate);
		printf("maxGRID Size %i %i %i\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("max Thread Dim %i %i %i \n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("max Threads per Block Dim  %i \n", prop.maxThreadsPerBlock);
		printf("total global memory %li \n", (long int)prop.totalGlobalMem *-1);
		printf("wrap size %i \n", prop.warpSize);
	
	}
}